#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>

#include "utils.hpp"
#include "common.hpp"
#include "cutil_subset.cuh"
#include "hip/hip_runtime_api.h"
#include "cuda_launch_config.cuh"

float* kmeans_cluster(size_t npoints, int dim, int nclusters,
                      const float *features, std::vector<int> &membership);
 
#define M 2
#define MAX_NUM_CLUSTERS 2048

__global__ void //__launch_bounds__(BLOCK_SIZE, 8)
IVFsearch(int K, int qsize, int dim, size_t npoints,
          const float *queries, 
          const float *data_vectors,
          int *results, 
          gpu_long_t* total_count_dc,
          int nclusters,
          const float* centroids,
          const int* clusters,
          const int* cluster_sizes,
          int max_cluster_size) {
  //int thread_id   = blockIdx.x * blockDim.x + threadIdx.x; // global thread index
  //int warp_id     = thread_id   / WARP_SIZE;               // global warp index
  //int num_warps   = (BLOCK_SIZE / WARP_SIZE) * gridDim.x;  // total number of active warps
  int thread_lane = threadIdx.x & (WARP_SIZE-1); // thread index within the warp
  int warp_lane   = threadIdx.x / WARP_SIZE;     // warp index within the CTA

  __shared__ uint64_t count_dc[WARPS_PER_BLOCK];
  __shared__ vidType candidates[BLOCK_SIZE*M];
  __shared__ float distances[BLOCK_SIZE*M];
  __shared__ vidType sorted_cids[MAX_NUM_CLUSTERS];
  __shared__ float c_dists[MAX_NUM_CLUSTERS];
  if (thread_lane == 0) count_dc[warp_lane] = 0;
  const int num_top_clusters = nclusters / 10;

  // for sorting
  typedef hipcub::BlockRadixSort<float, BLOCK_SIZE, M, vidType> BlockRadixSort;
  __shared__ typename BlockRadixSort::TempStorage temp_storage;

  int ROUNDS = (BLOCK_SIZE*M - K) / WARPS_PER_BLOCK;
  int NTASKS = ROUNDS * WARPS_PER_BLOCK;
  // each thread block takes a query
  for (int qid = blockIdx.x; qid < qsize; qid += gridDim.x) {
    const float *q_data = queries + qid * dim;

    // compute the distance between the query and centroids
    for (size_t cid = warp_lane; cid < nclusters; cid += WARPS_PER_BLOCK) {
      auto *c_data = centroids + cid * dim;
      auto dist = cutils::compute_distance(dim, q_data, c_data);
      if (thread_lane == 0) {
        count_dc[warp_lane] += 1;
        sorted_cids[cid] = cid;
        c_dists[cid] = dist;
      }
    }

    float thread_key[M];
    vidType thread_val[M];
    assert(MAX_NUM_CLUSTERS <= M*BLOCK_SIZE);
    // sort the centroids by distance, and pick the closest ones
    for (int j = 0; j < M; j++) {
      thread_key[j] = c_dists[j+M*threadIdx.x];
      thread_val[j] = sorted_cids[j+M*threadIdx.x];
    }
    BlockRadixSort(temp_storage).Sort(thread_key, thread_val);
    for (int j = 0; j < M; j++) {
      c_dists[j+M*threadIdx.x] = thread_key[j];
      sorted_cids[j+M*threadIdx.x] = thread_val[j];
    }
    __syncthreads();

    // start traversing the data points in the top clusters
    for (int i = 0; i < M; i++) {
      distances[BLOCK_SIZE*i+threadIdx.x] = FLT_MAX;
      candidates[BLOCK_SIZE*i+threadIdx.x] = BLOCK_SIZE*i+threadIdx.x;
    }
    __syncthreads();

    // the first cluster
    int cid = sorted_cids[0];
    auto cluster_0 = clusters + cid*max_cluster_size;
    assert(cluster_sizes[0] >= K);
    // insert the first K points in the first cluster
    for (size_t pid = warp_lane; pid < K; pid += WARPS_PER_BLOCK) {
      auto *p_data = data_vectors + cluster_0[pid] * dim;
      auto dist = cutils::compute_distance(dim, p_data, q_data);
      if (thread_lane == 0) {
        count_dc[warp_lane] += 1;
        distances[pid] = dist;
      }
    }
    __syncthreads();

    // each warp compares one point in the database
    for (size_t i = K+warp_lane; i < cluster_sizes[cid]; i += NTASKS) {
      for (int j = 0; j < ROUNDS; j++) {
        // in each rounds, every warp processes one data point
        auto pid = i + j * WARPS_PER_BLOCK;
        auto *p_data = data_vectors + cluster_0[pid] * dim;
        auto dist = cutils::compute_distance(dim, p_data, q_data);
        if (thread_lane == 0) {
          count_dc[warp_lane] += 1;
          distances[warp_lane+K+j*WARPS_PER_BLOCK] = dist;
          candidates[warp_lane+K+j*WARPS_PER_BLOCK] = pid;
        }
      }
      __syncthreads();

      // sort the queue by distance
      for (int j = 0; j < M; j++) {
        thread_key[j] = distances[j+M*threadIdx.x];
        thread_val[j] = candidates[j+M*threadIdx.x];
      }
      BlockRadixSort(temp_storage).Sort(thread_key, thread_val);
      for (int j = 0; j < M; j++) {
        distances[j+M*threadIdx.x] = thread_key[j];
        candidates[j+M*threadIdx.x] = thread_val[j];
      }
      __syncthreads();
    }

    // for the rest of the clusters
    for (int i = 1; i < num_top_clusters; i++) {
      int cid = sorted_cids[i];
      auto *cluster_i = clusters + cid*max_cluster_size;

      // each warp compares one point in the database
      for (size_t ii = warp_lane; ii < cluster_sizes[cid]; ii += NTASKS) {
        for (int j = 0; j < ROUNDS; j++) {
          // in each rounds, every warp processes one data point
          auto pid = i + j * WARPS_PER_BLOCK;
          auto *p_data = data_vectors + cluster_i[pid] * dim;
          auto dist = cutils::compute_distance(dim, p_data, q_data);
          if (thread_lane == 0) {
            count_dc[warp_lane] += 1;
            distances[warp_lane+K+j*WARPS_PER_BLOCK] = dist;
            candidates[warp_lane+K+j*WARPS_PER_BLOCK] = pid;
          }
        }
        __syncthreads();

        // sort the queue by distance
        for (int j = 0; j < M; j++) {
          thread_key[j] = distances[j+M*threadIdx.x];
          thread_val[j] = candidates[j+M*threadIdx.x];
        }
        BlockRadixSort(temp_storage).Sort(thread_key, thread_val);
        for (int j = 0; j < M; j++) {
          distances[j+M*threadIdx.x] = thread_key[j];
          candidates[j+M*threadIdx.x] = thread_val[j];
        }
        __syncthreads();
      }
    }

    // write the top-k elements into results
    for (int i = threadIdx.x; i < K; i += blockDim.x)
      results[qid * K + i] = candidates[i];
  }
  if (thread_lane == 0) atomicAdd(total_count_dc, count_dc[warp_lane]);
}

template <typename T>
void ANNS<T>::search(int k, int qsize, int dim, size_t npoints,
                     const T* queries, const T* data_vectors,
                     int *results, const char *index_file) {
  assert(npoints >= 10000);
  assert(K+WARPS_PER_BLOCK <= M*BLOCK_SIZE);
  size_t memsize = cutils::print_device_info(0);

  // clustering the data points
  int nclusters = std::sqrt(npoints);
  std::vector<int> membership(npoints, 0);
  auto centroids = kmeans_cluster(npoints, dim, nclusters, data_vectors, membership);
  std::vector<std::vector<int>> clusters(nclusters);
  for (size_t pt = 0; pt < npoints; ++pt) {
    auto cid = membership[pt];
    clusters[cid].push_back(pt);
  }
  std::vector<float> c_dist(nclusters);
  int max_cluster_size = 0;
  std::vector<int> cluster_sizes(nclusters);
  int cidx = 0;
  for (auto cluster : clusters) {
    cluster_sizes[cidx++] = cluster.size();
    if (cluster.size() > max_cluster_size)
      max_cluster_size = cluster.size();
  }

  // GPU lauch configuration
  size_t num_threads = BLOCK_SIZE;
  int max_blocks_per_SM = maximum_residency(IVFsearch, num_threads, 0);
  std::cout << "max_blocks_per_SM = " << max_blocks_per_SM << "\n";
  hipDeviceProp_t deviceProp;
  CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, 0));
  size_t num_blocks = max_blocks_per_SM * deviceProp.multiProcessorCount;
  assert(num_blocks < 65536);
  std::cout << "num_blocks = " << num_blocks << " num_threads = " << num_threads << "\n";

  // allocate device memory
  T* d_queries, *d_data, *d_centroids;
  CUDA_SAFE_CALL(hipMalloc((void **)&d_queries, qsize * dim * sizeof(T)));
  CUDA_SAFE_CALL(hipMemcpy(d_queries, queries, qsize * dim * sizeof(T), hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMalloc((void **)&d_data, npoints * dim * sizeof(T)));
  CUDA_SAFE_CALL(hipMemcpy(d_data, data_vectors, npoints * dim * sizeof(T), hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMalloc((void **)&d_centroids, nclusters * dim * sizeof(T)));
  CUDA_SAFE_CALL(hipMemcpy(d_centroids, centroids, nclusters * dim * sizeof(T), hipMemcpyHostToDevice));

  int *h_results = &results[0];
  int *d_results;
  CUDA_SAFE_CALL(hipMalloc((void **)&d_results, qsize * K * sizeof(int)));

  int *d_clusters, *d_cluster_sizes;
  CUDA_SAFE_CALL(hipMalloc((void **)&d_clusters, nclusters * max_cluster_size * sizeof(int)));
  for (int i = 0; i < nclusters; i++) {
    CUDA_SAFE_CALL(hipMemcpy(d_clusters + i * max_cluster_size, clusters[i].data(), clusters[i].size() * sizeof(int), hipMemcpyHostToDevice));
  }
  CUDA_SAFE_CALL(hipMalloc((void **)&d_cluster_sizes, nclusters * sizeof(int)));
  CUDA_SAFE_CALL(hipMemcpy(d_cluster_sizes, cluster_sizes.data(), nclusters * sizeof(int), hipMemcpyHostToDevice));

  gpu_long_t *d_total_count_dc;
  gpu_long_t total_count_dc = 0; 
  CUDA_SAFE_CALL(hipMalloc((void **)&d_total_count_dc, sizeof(gpu_long_t)));
  CUDA_SAFE_CALL(hipMemcpy(d_total_count_dc, &total_count_dc, sizeof(gpu_long_t), hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipDeviceSynchronize());

  //hipProfilerStart();
  Timer t;
  t.Start();
  IVFsearch<<<num_blocks, num_threads>>>(K, qsize, dim, npoints, 
                                         d_queries, d_data, d_results, 
                                         d_total_count_dc,
                                         nclusters, d_centroids, d_clusters, d_cluster_sizes, max_cluster_size);
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  t.Stop();
  //hipProfilerStop();

  double runtime = t.Seconds();
  auto throughput = double(qsize) / runtime;
  auto latency = runtime / qsize * 1000.0;
  std::cout << "runtime [brute_force_gpu] = " << runtime << " sec\n";
  std::cout << "throughput = " << throughput << " queries per second (QPS)\n";
  //printf("avg latency: %f ms/query\n", latency);
  CUDA_SAFE_CALL(hipMemcpy(h_results, d_results, qsize * K * sizeof(int), hipMemcpyDeviceToHost));
  CUDA_SAFE_CALL(hipMemcpy(&total_count_dc, d_total_count_dc, sizeof(gpu_long_t), hipMemcpyDeviceToHost));
  std::cout << "average # distance computation: " << npoints << "\n";

  CUDA_SAFE_CALL(hipFree(d_queries));
  CUDA_SAFE_CALL(hipFree(d_data));
  CUDA_SAFE_CALL(hipFree(d_results));
}

template class ANNS<float>;
