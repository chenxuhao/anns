#include "hip/hip_runtime.h"
#include "utils.hpp"
#include <hipcub/hipcub.hpp>
#include "graph_gpu.cuh"
#include "cutil_subset.cuh"
#include "hip/hip_runtime_api.h"
#include "cuda_launch_config.cuh"

// hyperparameters
#define BEAM_SIZE 128
#define M BEAM_SIZE/BLOCK_SIZE
#define MAX_DEG 32
#define NUM_START 64
#define LIMIT 1000000000

template <typename T>
__global__ void //__launch_bounds__(BLOCK_SIZE, 8)
BeamSearch(int K, int qsize, int dim, size_t npoints,
           const T* queries, const T* data_vectors,
           int *results,
           gpu_long_t* total_count_dc,
           vid_t *starting_points,
           GraphGPU<vid_t> g) {
  //int thread_id   = blockIdx.x * blockDim.x + threadIdx.x; // global thread index
  //int warp_id     = thread_id   / WARP_SIZE;               // global warp index
  int num_warps   = (BLOCK_SIZE / WARP_SIZE) * gridDim.x;  // total number of active warps
  int thread_lane = threadIdx.x & (WARP_SIZE-1); // thread index within the warp
  int warp_lane   = threadIdx.x / WARP_SIZE;     // warp index within the CTA
 
  __shared__ vid_t frontier[BEAM_SIZE];
  __shared__ vid_t unvisited_frontier[BEAM_SIZE];
  __shared__ vid_t visited[2*BEAM_SIZE];
  __shared__ vid_t new_frontier[BEAM_SIZE+MAX_DEG];
  __shared__ vid_t candidates[MAX_DEG];
  __shared__ float fro_dist[BEAM_SIZE];
  __shared__ float ufr_dist[BEAM_SIZE];
  __shared__ float visited_dist[2*BEAM_SIZE];
  __shared__ float nfr_dist[BEAM_SIZE+MAX_DEG];
  __shared__ float cand_dist[MAX_DEG];
  __shared__ uint64_t count_dc[WARPS_PER_BLOCK];
 
  // for sorting
  typedef hipcub::BlockRadixSort<float, BLOCK_SIZE, M, vid_t> BlockRadixSort;
  __shared__ typename BlockRadixSort::TempStorage temp_storage;

  // each thread block takes a query
  for (int qid = blockIdx.x; qid < qsize; qid += blockDim.x) {
    const float *q_data = queries + qid * dim;

    if (thread_lane == 0) count_dc[warp_lane] = 0;
    // insert nodes into frontier
    // each warp computes a distance
    for (int i = warp_lane; i < NUM_START; i += WARPS_PER_BLOCK) {
      auto v = starting_points[i];
      auto *v_data = data_vectors + v * dim;
      auto dist = cutils::compute_distance(dim, v_data, q_data);
      if (thread_lane == 0) {
        count_dc[warp_lane] += 1;
        frontier[i] = v;
        fro_dist[i] = dist;
      }
    }
    // sort frontier
    float thread_key[M];
    vid_t thread_val[M];
    for (int j = 0; j < M; j++) {
      thread_key[j] = frontier[j+M*threadIdx.x];
      thread_val[j] = fro_dist[j+M*threadIdx.x];
    }
    BlockRadixSort(temp_storage).Sort(thread_key, thread_val);
    for (int j = 0; j < M; j++) {
      frontier[j+M*threadIdx.x] = thread_key[j];
      fro_dist[j+M*threadIdx.x] = thread_val[j];
    }
    __syncthreads();

    if (threadIdx.x == 0) unvisited_frontier[0] = frontier[0];

    int remain = 1;
    int num_visited = 0;

    while (remain > 0 && num_visited < LIMIT) {
    }

    for (int i = thread_lane; i < K; i += WARP_SIZE) {
      results[qid * K + i] = candidates[i];
    }
  }
  if (thread_lane == 0) atomicAdd(total_count_dc, count_dc[warp_lane]);
}

template <typename T>
void ANNS<T>::search(int k, int qsize, int dim, size_t npoints,
                     const T* queries, const T* data_vectors,
                     int *results, const char *index_file) {
  size_t memsize = cutils::print_device_info(0);
  Graph<vid_t> g(index_file);
  assert(g.max_degree() <= MAX_DEG);

  // GPU lauch configuration
  size_t num_threads = BLOCK_SIZE;
  int max_blocks_per_SM = maximum_residency(BeamSearch<T>, num_threads, 0);
  std::cout << "max_blocks_per_SM = " << max_blocks_per_SM << "\n";
  hipDeviceProp_t deviceProp;
  CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, 0));
  size_t num_blocks = max_blocks_per_SM * deviceProp.multiProcessorCount;
  assert(num_blocks < 65536);
  std::cout << "num_blocks = " << num_blocks << " num_threads = " << num_threads << "\n";

  // allocate device memory
  T* d_queries, *d_data;
  CUDA_SAFE_CALL(hipMalloc((void **)&d_queries, qsize * dim * sizeof(T)));
  CUDA_SAFE_CALL(hipMemcpy(d_queries, queries, qsize * dim * sizeof(T), hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMalloc((void **)&d_data, npoints * dim * sizeof(T)));
  CUDA_SAFE_CALL(hipMemcpy(d_data, data_vectors, npoints * dim * sizeof(T), hipMemcpyHostToDevice));

  int *h_results = &results[0];
  int *d_results;
  CUDA_SAFE_CALL(hipMalloc((void **)&d_results, qsize * K * sizeof(int)));

  gpu_long_t *d_total_count_dc;
  gpu_long_t total_count_dc = 0; 
  CUDA_SAFE_CALL(hipMalloc((void **)&d_total_count_dc, sizeof(gpu_long_t)));
  CUDA_SAFE_CALL(hipMemcpy(d_total_count_dc, &total_count_dc, sizeof(gpu_long_t), hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipDeviceSynchronize());

  GraphGPU<vid_t> gg(g); // GPU graph 

  vid_t *starting_points = new vid_t[NUM_START];
  vid_t *d_starting_points;
  for (int i = 0; i < NUM_START; i++) starting_points[i] = rand() % npoints;
  CUDA_SAFE_CALL(hipMemcpy(d_starting_points, starting_points, NUM_START * sizeof(vid_t), hipMemcpyHostToDevice));

  //hipProfilerStart();
  Timer t;
  t.Start();
  BeamSearch<T><<<num_blocks, num_threads>>>(K, qsize, dim, npoints, 
                                          d_queries, d_data, d_results, 
                                          d_total_count_dc,
                                          d_starting_points,
                                          gg);
  t.Stop();
  //hipProfilerStop();

  double runtime = t.Seconds();
  auto throughput = double(qsize) / runtime;
  auto latency = runtime / qsize * 1000.0;
  std::cout << "runtime [brute_force_gpu] = " << runtime << " sec\n";
  std::cout << "throughput = " << throughput << " queries per second (QPS)\n";
  //printf("avg latency: %f ms/query\n", latency);
  CUDA_SAFE_CALL(hipMemcpy(h_results, d_results, qsize * K * sizeof(int), hipMemcpyDeviceToHost));
  CUDA_SAFE_CALL(hipMemcpy(&total_count_dc, d_total_count_dc, sizeof(gpu_long_t), hipMemcpyDeviceToHost));
  std::cout << "average # distance computation: " << npoints << "\n";

  CUDA_SAFE_CALL(hipFree(d_queries));
  CUDA_SAFE_CALL(hipFree(d_data));
  CUDA_SAFE_CALL(hipFree(d_results));
}

template class ANNS<float>;
